#include "hip/hip_runtime.h"
#include "tensor.h"
#include <cstddef>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <vector>

using namespace std;

extern "C" {
void GPUOperation::free_memory(vector<float> *data) {
    if (data != nullptr) {
        hipFree(data);
        data = nullptr;
    }
}

vector<float> *GPUOperation::move_data(vector<float> *data) {
    vector<float> *out = nullptr;
    if (data != nullptr) {
        auto size = data->size() * sizeof(float);
        hipMemcpy(out, data, size, hipMemcpyHostToDevice);
    }

    return out;
}

void GPUOperation::print_buffer(Buffer *buff) {
    auto out = string("Tensor(");
    for (size_t i = 0; i < buff->shape.at(0); i++) {
        if (i == 0) {
            out += "[";
        } else {
            out += "       [";
        }
        for (int j = 0; j < buff->shape[1]; j++) {
            out += to_string(buff->data->at(i * buff->shape[1] + j));
            if (j != buff->shape[1] - 1) {
                out += ", ";
            }
        }
        out += "]";
        if (i != buff->shape[0] - 1) {
            out += ",\n";
        }
    }
    cout << out << endl;
}

Buffer *GPUOperation::mul(Buffer *A, Buffer *B) { return nullptr; }

Buffer *GPUOperation::scalar_mul(Buffer *A, float c) { return nullptr; }

Buffer *GPUOperation::add(Buffer *A, Buffer *B) { return nullptr; }

Buffer *GPUOperation::subtract(Buffer *A, Buffer *B) { return nullptr; }

Buffer *GPUOperation::negate(Buffer *A) { return nullptr; }

Buffer *GPUOperation::pow(Buffer *A, float exp) { return nullptr; }

Buffer *GPUOperation::transpose(Buffer *data) { return nullptr; }

void BANANA() { cout << "BANANA" << endl; }
}