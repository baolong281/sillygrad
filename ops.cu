#include "hip/hip_runtime.h"
#include "tensor.h"
#include <hip/hip_runtime.h>

// Helper function to check CUDA errors
#define CHECK_CUDA(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": "     \
                << hipGetErrorString(err) << std::endl;                       \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)

// CUDA kernel for element-wise addition
__global__ void add_kernel(float *A, float *B, float *C, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    C[idx] = A[idx] + B[idx];
  }
}

// Implementation of GPU Operations
float* GPUOperation::move_data(float* cpu_data, size_t size) {
    float* gpu_data;
    CHECK_CUDA(hipMalloc(&gpu_data, size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(gpu_data, cpu_data, size * sizeof(float), hipMemcpyHostToDevice));
    return gpu_data;
}

void GPUOperation::free_memory(float* gpu_data) {
    if (gpu_data) {
        CHECK_CUDA(hipFree(gpu_data));
    }
}

Buffer* GPUOperation::add(Buffer* A, Buffer* B) {
    size_t size = A->shape[0] * A->shape[1];
    float* output;
    CHECK_CUDA(hipMalloc(&output, size * sizeof(float)));

    // Configure CUDA kernel
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;

    // Launch kernel
    add_kernel<<<num_blocks, block_size>>>(A->data, B->data, output, size);

    CHECK_CUDA(hipGetLastError());
    CHECK_CUDA(hipDeviceSynchronize());

    return new Buffer(output, "gpu", A->shape);
}
